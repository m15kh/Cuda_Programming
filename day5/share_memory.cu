
#include <hip/hip_runtime.h>
__global__ void simtExample(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N) {
        if(A[idx] > 0) {
            C[idx] = A[idx] + B[idx];
        } else {
            C[idx] = A[idx] - B[idx];
        }
    }
}

int main() {

}