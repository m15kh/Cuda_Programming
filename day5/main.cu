#include "hip/hip_runtime.h"
#include <iostream>  // Similar to "import" in Python for input/output

// This is a function that will run on the GPU
// __global__ means it's called from CPU but executes on GPU
__global__ void add(int *a, int *b, int *c, int N) {
    int index =  blockInx.x + blockDim.x + threadIdx.x
    if (index < N){
        c[index] = a[index] + b[index]
    }
}

int main() {

}