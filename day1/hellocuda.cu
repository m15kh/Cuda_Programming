
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU() {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello, World from GPU! Thread ID: %d\n", threadId);
}

int main() {
    // Launch the kernel
    helloFromGPU<<<3, 5>>>();
    
    // Wait for GPU to finish before accessing results

    hipDeviceSynchronize(); // ensures that the CPU waits for the GPU to finish executing all previously launched kernels 

    
    
    return 0;
}