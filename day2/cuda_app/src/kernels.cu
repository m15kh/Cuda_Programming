#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.h"

__global__ void dummykernel(){
    printf("Block %d, Thread %d\n", blockIdx.x, threadIdx.x);
}