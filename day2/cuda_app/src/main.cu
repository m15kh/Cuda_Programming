// filepath: src/main.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void dummykernel();

int main() {
    dummykernel<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}